#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>


// If the architecture is less than 600, it does not include the atomicAdd function and must define
#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double *address, double val)
{
    unsigned long long int *address_as_ull =
        (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

__global__ void pi_elem(double *values)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    values[i] = pow((double)-1, (double)i);
    values[i] /= (2 * i + 1);
}

int main()
{
    int max;
    double sum, sum2;
    std::string ent;
    max = 1000;

    std::cout << "Insert n iterations" << std::endl;

    std::cin >> ent;

    max = stoi(ent);
    
    // CUDA
    long N = (32 * max);
    double pi = 0;
    int i;
    double *d_values, *h_values;

    hipMalloc((void **)&d_values, N * sizeof(double));
    h_values = (double *)malloc(N * sizeof(double));
    auto startCuda = high_resolution_clock::now();
    pi_elem<<<N / 32, 32>>>(d_values);

    hipMemcpy(h_values, d_values, N * sizeof(double), hipMemcpyDeviceToHost);
    auto stopCuda = high_resolution_clock::now();
    printf("%f\n", h_values[0]);
    for (i = 0; i < N; i++)
    {
        pi += h_values[i];
    }
    printf("Aproximation: %f\n", 4 * pi);

    auto durationCuda = duration_cast<microseconds>(stopCuda - startCuda);
    std::cout << "Duracion:" << durationCuda.count() << "[uS]" << std::endl;

    return 0;
}